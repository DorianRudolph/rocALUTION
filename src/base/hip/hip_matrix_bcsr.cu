#include "hip/hip_runtime.h"
#include "../../utils/def.hpp"
#include "hip_matrix_csr.hpp"
#include "hip_matrix_bcsr.hpp"
#include "hip_vector.hpp"
#include "../host/host_matrix_bcsr.hpp"
#include "../base_matrix.hpp"
#include "../base_vector.hpp"
#include "../backend_manager.hpp"
#include "../../utils/log.hpp"
#include "hip_utils.hpp"
#include "hip_kernels_general.hpp"
#include "hip_kernels_bcsr.hpp"
#include "hip_allocate_free.hpp"
#include "../matrix_formats_ind.hpp"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_complex.h>

namespace paralution {

template <typename ValueType>
HIPAcceleratorMatrixBCSR<ValueType>::HIPAcceleratorMatrixBCSR() {

  // no default constructors
  LOG_INFO("no default constructor");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
HIPAcceleratorMatrixBCSR<ValueType>::HIPAcceleratorMatrixBCSR(const Paralution_Backend_Descriptor local_backend) {

  LOG_DEBUG(this, "HIPAcceleratorMatrixBCSR::HIPAcceleratorMatrixBCSR()",
            "constructor with local_backend");

  this->set_backend(local_backend); 

  CHECK_HIP_ERROR(__FILE__, __LINE__);

  // this is not working anyway...
  FATAL_ERROR(__FILE__, __LINE__);
}


template <typename ValueType>
HIPAcceleratorMatrixBCSR<ValueType>::~HIPAcceleratorMatrixBCSR() {

  LOG_DEBUG(this, "HIPAcceleratorMatrixBCSR::~HIPAcceleratorMatrixBCSR()",
            "destructor");

  this->Clear();

}

template <typename ValueType>
void HIPAcceleratorMatrixBCSR<ValueType>::info(void) const {

  LOG_INFO("HIPAcceleratorMatrixBCSR<ValueType>");

}

template <typename ValueType>
void HIPAcceleratorMatrixBCSR<ValueType>::AllocateBCSR(const int nnz, const int nrow, const int ncol) {

  assert(nnz >= 0);
  assert(ncol >= 0);
  assert(nrow >= 0);

  if (this->get_nnz() > 0)
    this->Clear();

  if (nnz > 0) {

    FATAL_ERROR(__FILE__, __LINE__);
   

  }

}

template <typename ValueType>
void HIPAcceleratorMatrixBCSR<ValueType>::Clear() {

  if (this->get_nnz() > 0) {

    FATAL_ERROR(__FILE__, __LINE__);


  }


}

template <typename ValueType>
void HIPAcceleratorMatrixBCSR<ValueType>::CopyFromHost(const HostMatrix<ValueType> &src) {

  const HostMatrixBCSR<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to HIP copy
  if ((cast_mat = dynamic_cast<const HostMatrixBCSR<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateBCSR(src.get_nnz(), src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    cast_mat->get_nnz();

    FATAL_ERROR(__FILE__, __LINE__);    
    
  } else {
    
    LOG_INFO("Error unsupported HIP matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void HIPAcceleratorMatrixBCSR<ValueType>::CopyToHost(HostMatrix<ValueType> *dst) const {

  HostMatrixBCSR<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // HIP to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixBCSR<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateBCSR(this->get_nnz(), this->get_nrow(), this->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    FATAL_ERROR(__FILE__, __LINE__);    
   
    
  } else {
    
    LOG_INFO("Error unsupported HIP matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void HIPAcceleratorMatrixBCSR<ValueType>::CopyFrom(const BaseMatrix<ValueType> &src) {

  const HIPAcceleratorMatrixBCSR<ValueType> *hip_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // HIP to HIP copy
  if ((hip_cast_mat = dynamic_cast<const HIPAcceleratorMatrixBCSR<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateBCSR(src.get_nnz(), src.get_nrow(), src.get_ncol() );  

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    hip_cast_mat->get_nnz();

    FATAL_ERROR(__FILE__, __LINE__);    

    
  } else {

    //CPU to HIP
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHost(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported HIP matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void HIPAcceleratorMatrixBCSR<ValueType>::CopyTo(BaseMatrix<ValueType> *dst) const {

  HIPAcceleratorMatrixBCSR<ValueType> *hip_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // HIP to HIP copy
  if ((hip_cast_mat = dynamic_cast<HIPAcceleratorMatrixBCSR<ValueType>*> (dst)) != NULL) {

    hip_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    hip_cast_mat->AllocateBCSR(dst->get_nnz(), dst->get_nrow(), dst->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    FATAL_ERROR(__FILE__, __LINE__);    
    
  } else {

    //HIP to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHost(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported HIP matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}


template <typename ValueType>
bool HIPAcceleratorMatrixBCSR<ValueType>::ConvertFrom(const BaseMatrix<ValueType> &mat) {

  this->Clear();

  // empty matrix is empty matrix
  if (mat.get_nnz() == 0)
    return true;


  const HIPAcceleratorMatrixBCSR<ValueType>   *cast_mat_bcsr;
  if ((cast_mat_bcsr = dynamic_cast<const HIPAcceleratorMatrixBCSR<ValueType>*> (&mat)) != NULL) {

      this->CopyFrom(*cast_mat_bcsr);
      return true;

  }

  /*
  const HIPAcceleratorMatrixCSR<ValueType>   *cast_mat_csr;
  if ((cast_mat_csr = dynamic_cast<const HIPAcceleratorMatrixCSR<ValueType>*> (&mat)) != NULL) {
    
    this->Clear();
    
    FATAL_ERROR(__FILE__, __LINE__);
    
    this->nrow_ = cast_mat_csr->get_nrow();
    this->ncol_ = cast_mat_csr->get_ncol();
    this->nnz_  = cast_mat_csr->get_nnz();
    
    return true;

  }
  */


  return false;

}

template <typename ValueType>
void HIPAcceleratorMatrixBCSR<ValueType>::Apply(const BaseVector<ValueType> &in, BaseVector<ValueType> *out) const {
/*
  assert(in.  get_size() >= 0);
  assert(out->get_size() >= 0);
  assert(in.  get_size() == this->get_ncol());
  assert(out->get_size() == this->get_nrow());


  const HIPAcceleratorVector<ValueType> *cast_in = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&in) ; 
  HIPAcceleratorVector<ValueType> *cast_out      = dynamic_cast<      HIPAcceleratorVector<ValueType>*> (out) ; 

  assert(cast_in != NULL);
  assert(cast_out!= NULL);
*/
  FATAL_ERROR(__FILE__, __LINE__);    

}


template <typename ValueType>
void HIPAcceleratorMatrixBCSR<ValueType>::ApplyAdd(const BaseVector<ValueType> &in, const ValueType scalar,
                                                  BaseVector<ValueType> *out) const {
  FATAL_ERROR(__FILE__, __LINE__);
}


template class HIPAcceleratorMatrixBCSR<double>;
template class HIPAcceleratorMatrixBCSR<float>;
#ifdef SUPPORT_COMPLEX
template class HIPAcceleratorMatrixBCSR<std::complex<double> >;
template class HIPAcceleratorMatrixBCSR<std::complex<float> >;
#endif

}
