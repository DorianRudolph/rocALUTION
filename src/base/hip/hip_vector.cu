#include "hip/hip_runtime.h"
#include "../../utils/def.hpp"
#include "hip_vector.hpp"
#include "../base_vector.hpp"
#include "../host/host_vector.hpp"
#include "../backend_manager.hpp"
#include "../../utils/log.hpp"
#include "../../utils/allocate_free.hpp"
#include "../../utils/math_functions.hpp"
#include "hip_utils.hpp"
#include "hip_kernels_general.hpp"
#include "hip_kernels_vector.hpp"
#include "hip_allocate_free.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipblas.h>

namespace paralution {

template <typename ValueType>
HIPAcceleratorVector<ValueType>::HIPAcceleratorVector() {

  // no default constructors
    LOG_INFO("no default constructor");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
HIPAcceleratorVector<ValueType>::HIPAcceleratorVector(const Paralution_Backend_Descriptor local_backend) {

  LOG_DEBUG(this, "HIPAcceleratorVector::HIPAcceleratorVector()",
            "constructor with local_backend");

  this->vec_ = NULL;
  this->set_backend(local_backend);

  this->index_array_  = NULL;
  this->index_buffer_ = NULL;

  this->host_buffer_ = NULL;
  this->device_buffer_ = NULL;

  CHECK_HIP_ERROR(__FILE__, __LINE__);

}


template <typename ValueType>
HIPAcceleratorVector<ValueType>::~HIPAcceleratorVector() {

  LOG_DEBUG(this, "HIPAcceleratorVector::~HIPAcceleratorVector()",
            "destructor");

  this->Clear();

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::info(void) const {

  LOG_INFO("HIPAcceleratorVector<ValueType>");

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::Allocate(const int n) {

  assert(n >= 0);

  if (this->get_size() >0)
    this->Clear();

  if (n > 0) {

    allocate_hip(n, &this->vec_);
    set_to_zero_hip(this->local_backend_.HIP_block_size, 
                    this->local_backend_.HIP_max_threads,
                    n, this->vec_);

    allocate_host(this->local_backend_.HIP_warp, &this->host_buffer_);
    allocate_hip(this->local_backend_.HIP_warp, &this->device_buffer_);

    this->size_ = n;
  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::SetDataPtr(ValueType **ptr, const int size) {

  assert(*ptr != NULL);
  assert(size > 0);

  hipDeviceSynchronize();

  this->vec_ = *ptr;
  this->size_ = size;

  allocate_host(this->local_backend_.HIP_warp, &this->host_buffer_);
  allocate_hip(this->local_backend_.HIP_warp, &this->device_buffer_);

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::LeaveDataPtr(ValueType **ptr) {

  assert(this->get_size() > 0);

  hipDeviceSynchronize();
  *ptr = this->vec_;
  this->vec_ = NULL;

  free_host(&this->host_buffer_);
  free_hip(&this->device_buffer_);

  this->size_ = 0;

}


template <typename ValueType>
void HIPAcceleratorVector<ValueType>::Clear(void) {
  
  if (this->get_size() > 0) {

    free_hip(&this->vec_);
    this->size_ = 0;

  }

  if (this->index_size_ > 0) {

    free_hip(&this->index_buffer_);
    free_hip(&this->index_array_);
    this->index_size_ = 0;

    free_host(&this->host_buffer_);
    free_hip(&this->device_buffer_);

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyFromHost(const HostVector<ValueType> &src) {

  // CPU to HIP copy
  const HostVector<ValueType> *cast_vec;
  if ((cast_vec = dynamic_cast<const HostVector<ValueType>*> (&src)) != NULL) {

    if (this->get_size() == 0) {

      // Allocate local structure
      this->Allocate(cast_vec->get_size());

      // Check for boundary
      assert(this->index_size_ == 0);
      if (cast_vec->index_size_ > 0) {

        this->index_size_ = cast_vec->index_size_;
        allocate_hip<int>(this->index_size_, &this->index_array_);
        allocate_hip<ValueType>(this->index_size_, &this->index_buffer_);

      }

    }

    assert(cast_vec->get_size() == this->get_size());
    assert(cast_vec->index_size_ == this->index_size_);

    if (this->get_size() > 0) {      

      hipblasStatus_t stat_t;
      stat_t = hipblasSetVector(this->get_size(), sizeof(ValueType),
                               cast_vec->vec_, // src
                               1,
                               this->vec_, // dst
                               1);
      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

      stat_t = hipblasSetVector(this->index_size_, sizeof(int),
                               cast_vec->index_array_,
                               1,
                               this->index_array_,
                               1);
      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    }

  } else {

    LOG_INFO("Error unsupported HIP vector type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyToHost(HostVector<ValueType> *dst) const {

  // HIP to CPU copy
  HostVector<ValueType> *cast_vec;
  if ((cast_vec = dynamic_cast<HostVector<ValueType>*> (dst)) != NULL) {

    if (cast_vec->get_size() == 0) {

      // Allocate local vector
      cast_vec->Allocate(this->get_size());

      // Check for boundary
      assert(cast_vec->index_size_ == 0);
      if (this->index_size_ > 0) {

        cast_vec->index_size_ = this->index_size_;
        allocate_host(this->index_size_, &cast_vec->index_array_);

      }

    }
      
    assert(cast_vec->get_size() == this->get_size());
    assert(cast_vec->index_size_ == this->index_size_);

    if (this->get_size() > 0) {

      hipblasStatus_t stat_t;
      stat_t = hipblasGetVector(this->get_size(), sizeof(ValueType),
                               this->vec_, // src
                               1,
                               cast_vec->vec_, // dst
                               1);
      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

      stat_t = hipblasGetVector(this->index_size_, sizeof(int),
                               this->index_array_,
                               1,
                               cast_vec->index_array_,
                               1);
      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    }

  } else {
    
    LOG_INFO("Error unsupported HIP vector type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

  
}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyFromHostAsync(const HostVector<ValueType> &src) {

  // CPU to HIP copy
  const HostVector<ValueType> *cast_vec;
  if ((cast_vec = dynamic_cast<const HostVector<ValueType>*> (&src)) != NULL) {

    if (this->get_size() == 0) {

      // Allocate local vector
      this->Allocate(cast_vec->get_size());

      // Check for boundary
      assert(this->index_size_ == 0);
      if (cast_vec->index_size_ > 0) {

        this->index_size_ = cast_vec->index_size_;
        allocate_hip<int>(this->index_size_, &this->index_array_);
        allocate_hip<ValueType>(this->index_size_, &this->index_buffer_);

      }

    }

    assert(cast_vec->get_size() == this->get_size());
    assert(cast_vec->index_size_ == this->index_size_);

    if (this->get_size() > 0) {

      hipMemcpyAsync(this->vec_,     // dst
                      cast_vec->vec_, // src
                      this->get_size()*sizeof(ValueType), // size
                      hipMemcpyHostToDevice);
      CHECK_HIP_ERROR(__FILE__, __LINE__);

      hipMemcpyAsync(this->index_array_,     // dst
                      cast_vec->index_array_, // src
                      this->index_size_*sizeof(int), // size
                      hipMemcpyHostToDevice);
      CHECK_HIP_ERROR(__FILE__, __LINE__);

    }

  } else {

    LOG_INFO("Error unsupported HIP vector type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyToHostAsync(HostVector<ValueType> *dst) const {

  // HIP to CPU copy
  HostVector<ValueType> *cast_vec;
  if ((cast_vec = dynamic_cast<HostVector<ValueType>*> (dst)) != NULL) {

    if (cast_vec->get_size() == 0) {

      // Allocate local vector
      cast_vec->Allocate(this->get_size());

      // Check for boundary
      assert(cast_vec->index_size_ == 0);
      if (this->index_size_ > 0) {

        cast_vec->index_size_ = this->index_size_;
        allocate_host(this->index_size_, &cast_vec->index_array_);

      }

    }

    assert(cast_vec->get_size() == this->get_size());
    assert(cast_vec->index_size_ == this->index_size_);

    if (this->get_size() > 0) {

      hipMemcpyAsync(cast_vec->vec_,  // dst
                      this->vec_,      // src
                      this->get_size()*sizeof(ValueType), // size
                      hipMemcpyDeviceToHost);
      CHECK_HIP_ERROR(__FILE__, __LINE__);

      hipMemcpyAsync(cast_vec->index_array_,  // dst
                      this->index_array_,      // src
                      this->index_size_*sizeof(int), // size
                      hipMemcpyDeviceToHost);
      CHECK_HIP_ERROR(__FILE__, __LINE__);

    }

  } else {

    LOG_INFO("Error unsupported HIP vector type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyFrom(const BaseVector<ValueType> &src) {

  const HIPAcceleratorVector<ValueType> *hip_cast_vec;
  const HostVector<ValueType> *host_cast_vec;

  // HIP to HIP copy
  if ((hip_cast_vec = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&src)) != NULL) {

    if (this->get_size() == 0) {

      // Allocate local vector
      this->Allocate(hip_cast_vec->get_size());

      // Check for boundary
      assert(this->index_size_ == 0);
      if (hip_cast_vec->index_size_ > 0) {

        this->index_size_ = hip_cast_vec->index_size_;
        allocate_hip<int>(this->index_size_, &this->index_array_);
        allocate_hip<ValueType>(this->index_size_, &this->index_buffer_);

      }

    }

    assert(hip_cast_vec->get_size() == this->get_size());
    assert(hip_cast_vec->index_size_ == this->index_size_);

    if (this != hip_cast_vec)  {  

      if (this->get_size() > 0) {

        hipMemcpy(this->vec_,         // dst
                   hip_cast_vec->vec_, // src
                   this->get_size()*sizeof(ValueType), // size
                   hipMemcpyDeviceToDevice);
        CHECK_HIP_ERROR(__FILE__, __LINE__);

        hipMemcpy(this->index_array_,            // dst
                   hip_cast_vec->index_array_,    // src
                   this->index_size_*sizeof(int), // size
                   hipMemcpyDeviceToDevice);
        CHECK_HIP_ERROR(__FILE__, __LINE__);

      }

    }

  } else {
    
    //HIP to CPU copy
    if ((host_cast_vec = dynamic_cast<const HostVector<ValueType>*> (&src)) != NULL) {
      

      this->CopyFromHost(*host_cast_vec);
      
    
    } else {

      LOG_INFO("Error unsupported HIP vector type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyFromAsync(const BaseVector<ValueType> &src) {

  const HIPAcceleratorVector<ValueType> *hip_cast_vec;
  const HostVector<ValueType> *host_cast_vec;

  // HIP to HIP copy
  if ((hip_cast_vec = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&src)) != NULL) {

    if (this->get_size() == 0) {

      // Allocate local vector
      this->Allocate(hip_cast_vec->get_size());

      // Check for boundary
      assert(this->index_size_ == 0);
      if (hip_cast_vec->index_size_ > 0) {

        this->index_size_ = hip_cast_vec->index_size_;
        allocate_hip<int>(this->index_size_, &this->index_array_);
        allocate_hip<ValueType>(this->index_size_, &this->index_buffer_);

      }

    }

    assert(hip_cast_vec->get_size() == this->get_size());
    assert(hip_cast_vec->index_size_ == this->index_size_);

    if (this != hip_cast_vec) {

      if (this->get_size() > 0) {

        hipMemcpy(this->vec_,         // dst
                   hip_cast_vec->vec_, // src
                   this->get_size()*sizeof(ValueType), // size
                   hipMemcpyDeviceToDevice);
        CHECK_HIP_ERROR(__FILE__, __LINE__);

        hipMemcpy(this->index_array_,         // dst
                   hip_cast_vec->index_array_, // src
                   this->index_size_*sizeof(int), // size
                   hipMemcpyDeviceToDevice);
        CHECK_HIP_ERROR(__FILE__, __LINE__);

      }

    }

  } else {

    //HIP to CPU copy
    if ((host_cast_vec = dynamic_cast<const HostVector<ValueType>*> (&src)) != NULL) {

      this->CopyFromHostAsync(*host_cast_vec);

    } else {

      LOG_INFO("Error unsupported HIP vector type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);

    }

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyFrom(const BaseVector<ValueType> &src,
                                               const int src_offset,
                                               const int dst_offset,
                                               const int size) {

  assert(&src != this);
  assert(this->get_size() > 0);
  assert(src.  get_size() > 0);
  assert(size > 0);

  assert(src_offset + size <= src.get_size());
  assert(dst_offset + size <= this->get_size());

  const HIPAcceleratorVector<ValueType> *cast_src = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&src);
  assert(cast_src != NULL);

  dim3 BlockSize(this->local_backend_.HIP_block_size);
  dim3 GridSize(size / this->local_backend_.HIP_block_size + 1);

  kernel_copy_offset_from<ValueType, int> <<<GridSize, BlockSize>>> (size, src_offset, dst_offset,
                                                                     cast_src->vec_, this->vec_);

  CHECK_HIP_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyTo(BaseVector<ValueType> *dst) const{

  HIPAcceleratorVector<ValueType> *hip_cast_vec;
  HostVector<ValueType> *host_cast_vec;

    // HIP to HIP copy
    if ((hip_cast_vec = dynamic_cast<HIPAcceleratorVector<ValueType>*> (dst)) != NULL) {

      if (hip_cast_vec->get_size() == 0) {

        // Allocate local vector
        hip_cast_vec->Allocate(this->get_size());

        // Check for boundary
        assert(hip_cast_vec->index_size_ == 0);
        if (this->index_size_ > 0) {

          hip_cast_vec->index_size_ = this->index_size_;
          allocate_hip<int>(this->index_size_, &hip_cast_vec->index_array_);
          allocate_hip<ValueType>(this->index_size_, &hip_cast_vec->index_buffer_);

        }

      }

      assert(hip_cast_vec->get_size() == this->get_size());
      assert(hip_cast_vec->index_size_ == this->index_size_);

      if (this != hip_cast_vec)  {

        if (this->get_size() >0) {

          hipMemcpy(hip_cast_vec->vec_, // dst
                     this->vec_,         // src
                     this->get_size()*sizeof(ValueType), // size
                     hipMemcpyDeviceToDevice);
          CHECK_HIP_ERROR(__FILE__, __LINE__);

          hipMemcpy(hip_cast_vec->index_array_,    // dst
                     this->index_array_,            // src
                     this->index_size_*sizeof(int), // size
                     hipMemcpyDeviceToDevice);
          CHECK_HIP_ERROR(__FILE__, __LINE__);

        }
      }

    } else {
      
      //HIP to CPU copy
      if ((host_cast_vec = dynamic_cast<HostVector<ValueType>*> (dst)) != NULL) {
        

        this->CopyToHost(host_cast_vec);
        
      
      } else {

        LOG_INFO("Error unsupported HIP vector type");
        this->info();
        dst->info();
        FATAL_ERROR(__FILE__, __LINE__);
        
      }
      
    }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyToAsync(BaseVector<ValueType> *dst) const {

  HIPAcceleratorVector<ValueType> *hip_cast_vec;
  HostVector<ValueType> *host_cast_vec;

  // HIP to HIP copy
  if ((hip_cast_vec = dynamic_cast<HIPAcceleratorVector<ValueType>*> (dst)) != NULL) {

    if (hip_cast_vec->get_size() == 0) {

      // Allocate local vector
      hip_cast_vec->Allocate(this->get_size());

      // Check for boundary
      assert(hip_cast_vec->index_size_ == 0);
      if (this->index_size_ > 0) {

        hip_cast_vec->index_size_ = this->index_size_;
        allocate_hip<int>(this->index_size_, &hip_cast_vec->index_array_);
        allocate_hip<ValueType>(this->index_size_, &hip_cast_vec->index_buffer_);

      }

    }

    assert(hip_cast_vec->get_size() == this->get_size());
    assert(hip_cast_vec->index_size_ == this->index_size_);

    if (this != hip_cast_vec) {

      if (this->get_size() > 0) {

        hipMemcpy(hip_cast_vec->vec_, // dst
                   this->vec_,         // src
                   this->get_size()*sizeof(ValueType), // size
                   hipMemcpyDeviceToDevice);
        CHECK_HIP_ERROR(__FILE__, __LINE__);

        hipMemcpy(hip_cast_vec->index_array_, // dst
                   this->index_array_,         // src
                   this->index_size_*sizeof(int), // size
                   hipMemcpyDeviceToDevice);
        CHECK_HIP_ERROR(__FILE__, __LINE__);

      }

    }

  } else {

    //HIP to CPU copy
    if ((host_cast_vec = dynamic_cast<HostVector<ValueType>*> (dst)) != NULL) {

      this->CopyToHostAsync(host_cast_vec);

    } else {

      LOG_INFO("Error unsupported HIP vector type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);

    }

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyFromFloat(const BaseVector<float> &src) {

  LOG_INFO("Mixed precision for non-complex to complex casting is not allowed");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <>
void HIPAcceleratorVector<double>::CopyFromFloat(const BaseVector<float> &src) {

  const HIPAcceleratorVector<float> *hip_cast_vec;

  // HIP to HIP copy
  if ((hip_cast_vec = dynamic_cast<const HIPAcceleratorVector<float>*> (&src)) != NULL) {

    if (this->get_size() == 0)
      this->Allocate(hip_cast_vec->get_size());

    assert(hip_cast_vec->get_size() == this->get_size());

    if (this->get_size() > 0) {

      dim3 BlockSize(this->local_backend_.HIP_block_size);
      dim3 GridSize(this->get_size() / this->local_backend_.HIP_block_size + 1);

      kernel_copy_from_float<double, int> <<<GridSize, BlockSize>>>(this->get_size(), hip_cast_vec->vec_, this->vec_);

      CHECK_HIP_ERROR(__FILE__, __LINE__);

    }

  } else {

    LOG_INFO("Error unsupported HIP vector type");
    FATAL_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyFromDouble(const BaseVector<double> &src) {

  LOG_INFO("Mixed precision for non-complex to complex casting is not allowed");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <>
void HIPAcceleratorVector<float>::CopyFromDouble(const BaseVector<double> &src) {

  const HIPAcceleratorVector<double> *hip_cast_vec;

  // HIP to HIP copy
  if ((hip_cast_vec = dynamic_cast<const HIPAcceleratorVector<double>*> (&src)) != NULL) {

    if (this->get_size() == 0)
      this->Allocate(hip_cast_vec->get_size());

    assert(hip_cast_vec->get_size() == this->get_size());

    if (this->get_size()  >0) {

      dim3 BlockSize(this->local_backend_.HIP_block_size);
      dim3 GridSize(this->get_size() / this->local_backend_.HIP_block_size + 1);

      kernel_copy_from_double<float, int> <<<GridSize, BlockSize>>>(this->get_size(), hip_cast_vec->vec_, this->vec_);

      CHECK_HIP_ERROR(__FILE__, __LINE__);
    }

  } else {
    LOG_INFO("Error unsupported HIP vector type");
    FATAL_ERROR(__FILE__, __LINE__);

  }
  
}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyFromData(const ValueType *data) {

  if (this->get_size() > 0) {

    hipMemcpy(this->vec_,                         // dst
               data,                               // src
               this->get_size()*sizeof(ValueType), // size
               hipMemcpyDeviceToDevice);
    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyToData(ValueType *data) const {

  if (this->get_size() > 0) {

    hipMemcpy(data,                               // dst
               this->vec_,                         // src
               this->get_size()*sizeof(ValueType), // size
               hipMemcpyDeviceToDevice);
    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::Zeros(void) {

  if (this->get_size() > 0) {

    set_to_zero_hip(this->local_backend_.HIP_block_size,
                    this->local_backend_.HIP_max_threads,
                    this->get_size(), this->vec_);

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::Ones(void) {

  if (this->get_size() > 0)
    set_to_one_hip(this->local_backend_.HIP_block_size, 
                   this->local_backend_.HIP_max_threads,
                   this->get_size(), this->vec_);

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::SetValues(const ValueType val) {

  LOG_INFO("HIPAcceleratorVector::SetValues NYI");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <>
void HIPAcceleratorVector<double>::AddScale(const BaseVector<double> &x, const double alpha) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());
    
    const HIPAcceleratorVector<double> *cast_x = dynamic_cast<const HIPAcceleratorVector<double>*> (&x);
    assert(cast_x != NULL);
    
    hipblasStatus_t stat_t;
    
    stat_t = hipblasDaxpy(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle), 
                         this->get_size(), 
                         &alpha, 
                         cast_x->vec_, 1,
                         this->vec_, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

}

template <>
void HIPAcceleratorVector<float>::AddScale(const BaseVector<float> &x, const float alpha) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());
    
    const HIPAcceleratorVector<float> *cast_x = dynamic_cast<const HIPAcceleratorVector<float>*> (&x);
    assert(cast_x != NULL);
    
    hipblasStatus_t stat_t;
    
    stat_t = hipblasSaxpy(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle), 
                         this->get_size(), 
                         &alpha, 
                         cast_x->vec_, 1,
                         this->vec_, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

}

template <>
void HIPAcceleratorVector<std::complex<double> >::AddScale(const BaseVector<std::complex<double> > &x,
                                                           const std::complex<double> alpha) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());

    const HIPAcceleratorVector<std::complex<double> > *cast_x = dynamic_cast<const HIPAcceleratorVector<std::complex<double> >*> (&x);
    assert(cast_x != NULL);

    hipblasStatus_t stat_t;

    stat_t = hipblasZaxpy(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle), 
                         this->get_size(),
                         (hipDoubleComplex*)&alpha,
                         (hipDoubleComplex*)cast_x->vec_, 1,
                         (hipDoubleComplex*)this->vec_, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

}

template <>
void HIPAcceleratorVector<std::complex<float> >::AddScale(const BaseVector<std::complex<float> > &x,
                                                          const std::complex<float> alpha) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());

    const HIPAcceleratorVector<std::complex<float> > *cast_x = dynamic_cast<const HIPAcceleratorVector<std::complex<float> >*> (&x);
    assert(cast_x != NULL);

    hipblasStatus_t stat_t;

    stat_t = hipblasCaxpy(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle), 
                         this->get_size(),
                         (hipFloatComplex*)&alpha,
                         (hipFloatComplex*)cast_x->vec_, 1,
                         (hipFloatComplex*)this->vec_, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

}

template <>
void HIPAcceleratorVector<int>::AddScale(const BaseVector<int> &x, const int alpha) {

  LOG_INFO("No int CUBLAS axpy function");
  FATAL_ERROR(__FILE__, __LINE__);
 
}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::ScaleAdd(const ValueType alpha, const BaseVector<ValueType> &x) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());

    const HIPAcceleratorVector<ValueType> *cast_x = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&x);
    assert(cast_x != NULL);

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.HIP_block_size);
    dim3 GridSize(size / this->local_backend_.HIP_block_size + 1);

    kernel_scaleadd<<<GridSize, BlockSize>>> (size, HIPVal(alpha), HIPPtr(cast_x->vec_), HIPPtr(this->vec_));

    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::ScaleAddScale(const ValueType alpha, const BaseVector<ValueType> &x, const ValueType beta) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());

    const HIPAcceleratorVector<ValueType> *cast_x = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&x);
    assert(cast_x != NULL);

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.HIP_block_size);
    dim3 GridSize(size / this->local_backend_.HIP_block_size + 1);

    kernel_scaleaddscale<<<GridSize, BlockSize>>> (size, HIPVal(alpha), HIPVal(beta), HIPPtr(cast_x->vec_), HIPPtr(this->vec_));

    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::ScaleAddScale(const ValueType alpha, const BaseVector<ValueType> &x, const ValueType beta,
                                                    const int src_offset, const int dst_offset,const int size) {

  if (this->get_size() > 0) {

    assert(this->get_size() > 0);
    assert(x.get_size() > 0);
    assert(size > 0);
    assert(src_offset + size <= x.get_size());
    assert(dst_offset + size <= this->get_size());

    const HIPAcceleratorVector<ValueType> *cast_x = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&x);
    assert(cast_x != NULL);

    dim3 BlockSize(this->local_backend_.HIP_block_size);
    dim3 GridSize(size / this->local_backend_.HIP_block_size + 1);

    kernel_scaleaddscale_offset<<<GridSize, BlockSize>>> (size, src_offset, dst_offset,
                                                          HIPVal(alpha), HIPVal(beta),
                                                          HIPPtr(cast_x->vec_), HIPPtr(this->vec_));
    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::ScaleAdd2(const ValueType alpha, const BaseVector<ValueType> &x,
                                                const ValueType beta, const BaseVector<ValueType> &y,
                                                const ValueType gamma) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());
    assert(this->get_size() == y.get_size());

    const HIPAcceleratorVector<ValueType> *cast_x = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&x);
    const HIPAcceleratorVector<ValueType> *cast_y = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&y);
    assert(cast_x != NULL);
    assert(cast_y != NULL);

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.HIP_block_size);
    dim3 GridSize(size / this->local_backend_.HIP_block_size + 1);

    kernel_scaleadd2<<<GridSize, BlockSize>>> (size, HIPVal(alpha), HIPVal(beta), HIPVal(gamma),
                                               HIPPtr(cast_x->vec_), HIPPtr(cast_y->vec_), HIPPtr(this->vec_));
    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

}

template <>
void HIPAcceleratorVector<double>::Scale(const double alpha) {

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasDscal(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(), &alpha,
                         this->vec_, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

}

template <>
void HIPAcceleratorVector<float>::Scale(const float alpha) {

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasSscal(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(), &alpha,
                         this->vec_, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

}

template <>
void HIPAcceleratorVector<std::complex<double> >::Scale(const std::complex<double> alpha) {

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasZscal(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(), (hipDoubleComplex*)&alpha,
                         (hipDoubleComplex*)this->vec_, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

}

template <>
void HIPAcceleratorVector<std::complex<float> >::Scale(const std::complex<float> alpha) {

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasCscal(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(), (hipFloatComplex*)&alpha,
                         (hipFloatComplex*)this->vec_, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

}

template <>
void HIPAcceleratorVector<int>::Scale(const int alpha) {

  LOG_INFO("No int CUBLAS scale function");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::ExclusiveScan(const BaseVector<ValueType> &x) {

  LOG_INFO("HIPAcceleratorVector::ExclusiveScan() NYI");
  FATAL_ERROR(__FILE__, __LINE__); 

}

template <>
double HIPAcceleratorVector<double>::Dot(const BaseVector<double> &x) const {

  assert(this->get_size() == x.get_size());

  const HIPAcceleratorVector<double> *cast_x = dynamic_cast<const HIPAcceleratorVector<double>*> (&x);
  assert(cast_x != NULL);

  double res = 0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasDdot(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                        this->get_size(),
                        this->vec_, 1,
                        cast_x->vec_, 1, &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
float HIPAcceleratorVector<float>::Dot(const BaseVector<float> &x) const {

  assert(this->get_size() == x.get_size());

  const HIPAcceleratorVector<float> *cast_x = dynamic_cast<const HIPAcceleratorVector<float>*> (&x);
  assert(cast_x != NULL);

  float res = 0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasSdot(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                        this->get_size(),
                        this->vec_, 1,
                        cast_x->vec_, 1, &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
std::complex<double> HIPAcceleratorVector<std::complex<double> >::Dot(const BaseVector<std::complex<double> > &x) const {

  assert(this->get_size() == x.get_size());

  const HIPAcceleratorVector<std::complex<double> > *cast_x = dynamic_cast<const HIPAcceleratorVector<std::complex<double> >*> (&x);
  assert(cast_x != NULL);

  std::complex<double> res = std::complex<double>(double(0.0), double(0.0));

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasZdotc(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(),
                         (hipDoubleComplex*)this->vec_, 1,
                         (hipDoubleComplex*)cast_x->vec_, 1, (hipDoubleComplex*)&res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
std::complex<float> HIPAcceleratorVector<std::complex<float> >::Dot(const BaseVector<std::complex<float> > &x) const {

  assert(this->get_size() == x.get_size());

  const HIPAcceleratorVector<std::complex<float> > *cast_x = dynamic_cast<const HIPAcceleratorVector<std::complex<float> >*> (&x);
  assert(cast_x != NULL);

  std::complex<float> res = std::complex<float>(float(0.0), float(0.0));

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasCdotc(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(),
                         (hipFloatComplex*)this->vec_, 1,
                         (hipFloatComplex*)cast_x->vec_, 1, (hipFloatComplex*)&res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
int HIPAcceleratorVector<int>::Dot(const BaseVector<int> &x) const {

  LOG_INFO("No int CUBLAS dot function");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
ValueType HIPAcceleratorVector<ValueType>::DotNonConj(const BaseVector<ValueType> &x) const {

  return this->Dot(x);

}

template <>
std::complex<double> HIPAcceleratorVector<std::complex<double> >::DotNonConj(const BaseVector<std::complex<double> > &x) const {

  assert(this->get_size() == x.get_size());

  const HIPAcceleratorVector<std::complex<double> > *cast_x = dynamic_cast<const HIPAcceleratorVector<std::complex<double> >*> (&x);
  assert(cast_x != NULL);

  std::complex<double> res = std::complex<double>(double(0.0), double(0.0));

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasZdotu(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(),
                         (hipDoubleComplex*)this->vec_, 1,
                         (hipDoubleComplex*)cast_x->vec_, 1, (hipDoubleComplex*)&res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
std::complex<float> HIPAcceleratorVector<std::complex<float> >::DotNonConj(const BaseVector<std::complex<float> > &x) const {

  assert(this->get_size() == x.get_size());

  const HIPAcceleratorVector<std::complex<float> > *cast_x = dynamic_cast<const HIPAcceleratorVector<std::complex<float> >*> (&x);
  assert(cast_x != NULL);

  std::complex<float> res = std::complex<float>(float(0.0), float(0.0));

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasCdotu(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(),
                         (hipFloatComplex*)this->vec_, 1,
                         (hipFloatComplex*)cast_x->vec_, 1, (hipFloatComplex*)&res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
int HIPAcceleratorVector<int>::DotNonConj(const BaseVector<int> &x) const {

  LOG_INFO("No int CUBLAS dot function");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <>
double HIPAcceleratorVector<double>::Norm(void) const {

  double res = 0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasDnrm2(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(),
                         this->vec_, 1, &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
float HIPAcceleratorVector<float>::Norm(void) const {

  float res = 0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasSnrm2(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(),
                         this->vec_, 1, &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
std::complex<double> HIPAcceleratorVector<std::complex<double> >::Norm(void) const {

  double res = double(0.0);

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasDznrm2(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                          this->get_size(),
                          (hipDoubleComplex*)this->vec_, 1, &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  std::complex<double> c_res = (std::complex<double>) res;
  return c_res;

}

template <>
std::complex<float> HIPAcceleratorVector<std::complex<float> >::Norm(void) const {

  float res = float(0.0);

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasScnrm2(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                          this->get_size(),
                          (hipFloatComplex*)this->vec_, 1, &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  std::complex<float> c_res = (std::complex<float>) res;
  return c_res;

}

template <>
int HIPAcceleratorVector<int>::Norm(void) const {

  LOG_INFO("What is int HIPAcceleratorVector<ValueType>::Norm(void) const?");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
ValueType HIPAcceleratorVector<ValueType>::Reduce(void) const {

  ValueType res = (ValueType) 0;

  if (this->get_size() > 0) {

    reduce_hip<int, ValueType, 32, 256>(this->get_size(), this->vec_, &res, this->host_buffer_, this->device_buffer_);
    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

  return res;

}

template <>
int HIPAcceleratorVector<int>::Reduce(void) const {

  LOG_INFO("Reduce<int> not implemented");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <>
double HIPAcceleratorVector<double>::Asum(void) const {

  double res = 0.0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasDasum(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(),
                         this->vec_, 1,
                         &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
float HIPAcceleratorVector<float>::Asum(void) const {

  float res = 0.0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasSasum(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(),
                         this->vec_, 1,
                         &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
std::complex<double> HIPAcceleratorVector<std::complex<double> >::Asum(void) const {

  double res = double(0.0);

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasDzasum(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(),
                         (hipDoubleComplex*)this->vec_, 1,
                         &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
std::complex<float> HIPAcceleratorVector<std::complex<float> >::Asum(void) const {

  float res = float(0.0);

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasScasum(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                         this->get_size(),
                         (hipFloatComplex*)this->vec_, 1,
                         &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
int HIPAcceleratorVector<int>::Asum(void) const {

  LOG_INFO("Asum<int> not implemented");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <>
int HIPAcceleratorVector<double>::Amax(double &value) const {

  int index = 0;
  value = 0.0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasIdamax(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                          this->get_size(),
                          this->vec_, 1, &index);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    // cublas returns 1-based indexing
    --index;

    hipMemcpy(&value,
               this->vec_+index,
               sizeof(double),
               hipMemcpyDeviceToHost);
    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

  value = paralution_abs(value);
  return index;

}

template <>
int HIPAcceleratorVector<float>::Amax(float &value) const {

  int index = 0;
  value = 0.0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasIsamax(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                          this->get_size(),
                          this->vec_, 1, &index);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    // cublas returns 1-based indexing
    --index;

    hipMemcpy(&value,
               this->vec_+index,
               sizeof(float),
               hipMemcpyDeviceToHost);
    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

  value = paralution_abs(value);
  return index;

}

template <>
int HIPAcceleratorVector<std::complex<double> >::Amax(std::complex<double> &value) const {

  int index = 0;
  value = std::complex<double>(double(0.0), double(0.0));

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasIzamax(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                          this->get_size(),
                          (hipDoubleComplex*)this->vec_, 1, &index);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    // cublas returns 1-based indexing
    --index;

    hipMemcpy(&value,
               (hipDoubleComplex*)this->vec_+index,
               sizeof(std::complex<double>),
               hipMemcpyDeviceToHost);
    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

  value = paralution_abs(value);
  return index;

}

template <>
int HIPAcceleratorVector<std::complex<float> >::Amax(std::complex<float> &value) const {

  int index = 0;
  value = std::complex<float>(float(0.0), float(0.0));

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasIcamax(CUBLAS_HANDLE(this->local_backend_.HIP_rocblas_handle),
                          this->get_size(),
                          (hipFloatComplex*)this->vec_, 1, &index);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    // cublas returns 1-based indexing
    --index;

    hipMemcpy(&value,
               (hipFloatComplex*)this->vec_+index,
               sizeof(std::complex<float>),
               hipMemcpyDeviceToHost);
    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

  value = paralution_abs(value);
  return index;

}

template <>
int HIPAcceleratorVector<int>::Amax(int &value) const {

  LOG_INFO("Amax<int> not implemented");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::PointWiseMult(const BaseVector<ValueType> &x) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());

    const HIPAcceleratorVector<ValueType> *cast_x = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&x);
    assert(cast_x != NULL);

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.HIP_block_size);
    dim3 GridSize(size / this->local_backend_.HIP_block_size + 1);

    kernel_pointwisemult<<<GridSize, BlockSize>>> (size, HIPPtr(cast_x->vec_), HIPPtr(this->vec_));

    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::PointWiseMult(const BaseVector<ValueType> &x, const BaseVector<ValueType> &y) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());
    assert(this->get_size() == y.get_size());

    const HIPAcceleratorVector<ValueType> *cast_x = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&x);
    const HIPAcceleratorVector<ValueType> *cast_y = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&y);
    assert(cast_x != NULL);
    assert(cast_y != NULL);

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.HIP_block_size);
    dim3 GridSize(size / this->local_backend_.HIP_block_size + 1);

    kernel_pointwisemult2<<<GridSize, BlockSize>>> (size, HIPPtr(cast_x->vec_), HIPPtr(cast_y->vec_), HIPPtr(this->vec_));

    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::Permute(const BaseVector<int> &permutation) {

  if (this->get_size() > 0) {

    assert(this->get_size() == permutation.get_size());
    
    const HIPAcceleratorVector<int> *cast_perm = dynamic_cast<const HIPAcceleratorVector<int>*> (&permutation);
    assert(cast_perm != NULL);
    
    HIPAcceleratorVector<ValueType> vec_tmp(this->local_backend_);     
    vec_tmp.Allocate(this->get_size());
    vec_tmp.CopyFrom(*this);
    
    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.HIP_block_size);
    dim3 GridSize(size / this->local_backend_.HIP_block_size + 1);
    
    //    this->vec_[ cast_perm->vec_[i] ] = vec_tmp.vec_[i];  
    kernel_permute<ValueType, int> <<<GridSize, BlockSize>>> (size, cast_perm->vec_, vec_tmp.vec_, this->vec_);
    
    CHECK_HIP_ERROR(__FILE__, __LINE__);      
  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::PermuteBackward(const BaseVector<int> &permutation) {

  if (this->get_size() > 0) {

    assert(this->get_size() == permutation.get_size());
    
    const HIPAcceleratorVector<int> *cast_perm = dynamic_cast<const HIPAcceleratorVector<int>*> (&permutation);
    assert(cast_perm != NULL);
    
    HIPAcceleratorVector<ValueType> vec_tmp(this->local_backend_);   
    vec_tmp.Allocate(this->get_size());
    vec_tmp.CopyFrom(*this);
    
    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.HIP_block_size);
    dim3 GridSize(size / this->local_backend_.HIP_block_size + 1);
    
    //    this->vec_[i] = vec_tmp.vec_[ cast_perm->vec_[i] ];
    kernel_permute_backward<ValueType, int> <<<GridSize, BlockSize>>> (size, cast_perm->vec_, vec_tmp.vec_, this->vec_);
    
    CHECK_HIP_ERROR(__FILE__, __LINE__);      
  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyFromPermute(const BaseVector<ValueType> &src,
                                                      const BaseVector<int> &permutation) { 

  if (this->get_size() > 0) {

    assert(this != &src);
    
    const HIPAcceleratorVector<ValueType> *cast_vec = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&src);
    const HIPAcceleratorVector<int> *cast_perm      = dynamic_cast<const HIPAcceleratorVector<int>*> (&permutation) ; 
    assert(cast_perm != NULL);
    assert(cast_vec  != NULL);
    
    assert(cast_vec ->get_size() == this->get_size());
    assert(cast_perm->get_size() == this->get_size());
    
    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.HIP_block_size);
    dim3 GridSize(size / this->local_backend_.HIP_block_size + 1);
    
    //    this->vec_[ cast_perm->vec_[i] ] = cast_vec->vec_[i];
    kernel_permute<ValueType, int> <<<GridSize, BlockSize>>> (size, cast_perm->vec_, cast_vec->vec_, this->vec_);
    
    CHECK_HIP_ERROR(__FILE__, __LINE__);      
  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::CopyFromPermuteBackward(const BaseVector<ValueType> &src,
                                                              const BaseVector<int> &permutation) {

  if (this->get_size() > 0) {

    assert(this != &src);
    
    const HIPAcceleratorVector<ValueType> *cast_vec = dynamic_cast<const HIPAcceleratorVector<ValueType>*> (&src);
    const HIPAcceleratorVector<int> *cast_perm      = dynamic_cast<const HIPAcceleratorVector<int>*> (&permutation) ; 
    assert(cast_perm != NULL);
    assert(cast_vec  != NULL);
    
    assert(cast_vec ->get_size() == this->get_size());
    assert(cast_perm->get_size() == this->get_size());
    
    
    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.HIP_block_size);
    dim3 GridSize(size / this->local_backend_.HIP_block_size + 1);
    
    //    this->vec_[i] = cast_vec->vec_[ cast_perm->vec_[i] ];
    kernel_permute_backward<ValueType, int> <<<GridSize, BlockSize>>> (size, cast_perm->vec_, cast_vec->vec_, this->vec_);
    
    CHECK_HIP_ERROR(__FILE__, __LINE__);      
  }

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::SetIndexArray(const int size, const int *index) {

  assert(size > 0);
  assert(this->get_size() >= size);

  this->index_size_ = size;

  allocate_hip<int>(this->index_size_, &this->index_array_);
  allocate_hip<ValueType>(this->index_size_, &this->index_buffer_);

  hipMemcpy(this->index_array_,            // dst
             index,                         // src
             this->index_size_*sizeof(int), // size
             hipMemcpyHostToDevice);

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::GetIndexValues(ValueType *values) const {

  assert(values != NULL);

  dim3 BlockSize(this->local_backend_.HIP_block_size);
  dim3 GridSize(this->index_size_ / this->local_backend_.HIP_block_size + 1);

  kernel_get_index_values<ValueType, int> <<<GridSize, BlockSize>>> (this->index_size_, this->index_array_,
                                                                     this->vec_, this->index_buffer_);
  CHECK_HIP_ERROR(__FILE__, __LINE__);

  hipMemcpy(values,                              // dst
             this->index_buffer_,                 // src
             this->index_size_*sizeof(ValueType), // size
             hipMemcpyDeviceToHost);
  CHECK_HIP_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::SetIndexValues(const ValueType *values) {

  assert(values != NULL);

  hipMemcpy(this->index_buffer_,
             values,
             this->index_size_*sizeof(ValueType),
             hipMemcpyHostToDevice);
  CHECK_HIP_ERROR(__FILE__, __LINE__);

  dim3 BlockSize(this->local_backend_.HIP_block_size);
  dim3 GridSize(this->index_size_ / this->local_backend_.HIP_block_size + 1);

  kernel_set_index_values<ValueType, int> <<<GridSize, BlockSize>>> (this->index_size_, this->index_array_,
                                                                     this->index_buffer_, this->vec_);
  CHECK_HIP_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::GetContinuousValues(const int start, const int end, ValueType *values) const {

  assert(start >= 0);
  assert(end >= start);
  assert(end <= this->get_size());
  assert(values != NULL);

  hipMemcpy(values,                        // dst
             this->vec_+start,              // src
             (end-start)*sizeof(ValueType), // size
             hipMemcpyDeviceToHost);
  CHECK_HIP_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::SetContinuousValues(const int start, const int end, const ValueType *values) {

  assert(start >= 0);
  assert(end >= start);
  assert(end <= this->get_size());
  assert(values != NULL);

  hipMemcpy(this->vec_+start,              // dst
             values,                        // src
             (end-start)*sizeof(ValueType), // size
             hipMemcpyHostToDevice);
  CHECK_HIP_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::ExtractCoarseMapping(const int start, const int end, const int *index,
                                                           const int nc, int *size, int *map) const {

  LOG_INFO("ExtractCoarseMapping() NYI for HIP");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
void HIPAcceleratorVector<ValueType>::ExtractCoarseBoundary(const int start, const int end, const int *index,
                                                            const int nc, int *size, int *boundary) const {

  LOG_INFO("ExtractCoarseBoundary() NYI for HIP");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <>
void HIPAcceleratorVector<double>::Power(const double power) {

  if (this->get_size() > 0) {

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.HIP_block_size);
    dim3 GridSize(size / this->local_backend_.HIP_block_size + 1);

    kernel_powerd<int> <<<GridSize, BlockSize>>> (size, power, this->vec_);

    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

}

template <>
void HIPAcceleratorVector<float>::Power(const double power) {

  if (this->get_size() > 0) {

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.HIP_block_size);
    dim3 GridSize(size / this->local_backend_.HIP_block_size + 1);

    kernel_powerf<int> <<<GridSize, BlockSize>>> (size, power, this->vec_);

    CHECK_HIP_ERROR(__FILE__, __LINE__);

  }

}

template <>
void HIPAcceleratorVector<std::complex<double> >::Power(const double power) {

  if (this->get_size() > 0) {

    LOG_INFO("HIPAcceleratorVector::Power(), no pow() for std::complex<double> in HIP");
    FATAL_ERROR(__FILE__, __LINE__);

  }

}

template <>
void HIPAcceleratorVector<std::complex<float> >::Power(const double power) {

  if (this->get_size() > 0) {

    LOG_INFO("HIPAcceleratorVector::Power(), no pow() for std::complex<float> in HIP");
    FATAL_ERROR(__FILE__, __LINE__);

  }

}

template <>
void HIPAcceleratorVector<int>::Power(const double power) {

  if (this->get_size() > 0) {

    LOG_INFO("HIPAcceleratorVector::Power(), no pow() for int in HIP");
    FATAL_ERROR(__FILE__, __LINE__);


  }

}


template class HIPAcceleratorVector<double>;
template class HIPAcceleratorVector<float>;
#ifdef SUPPORT_COMPLEX
template class HIPAcceleratorVector<std::complex<double> >;
template class HIPAcceleratorVector<std::complex<float> >;
#endif
template class HIPAcceleratorVector<int>;

}
